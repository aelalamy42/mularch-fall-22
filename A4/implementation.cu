/*
============================================================================
Filename    : algorithm.c
Author      : Elalamy Balducci
SCIPER      : ______ - 325035
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}

__global__
void single_iteration(double* input, double* output, int length){
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    // Offset of one not to touch the border of the matrix
    // As we only have length - 2 threads in each dimension, the max is by definition length - 1
    output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                        input[(i-1)*(length)+(j)]   +
                                        input[(i-1)*(length)+(j+1)] +
                                        input[(i)*(length)+(j-1)]   +
                                        input[(i)*(length)+(j)]     +
                                        input[(i)*(length)+(j+1)]   +
                                        input[(i+1)*(length)+(j-1)] +
                                        input[(i+1)*(length)+(j)]   +
                                        input[(i+1)*(length)+(j+1)] ) / 9;
}

__global__
void init_center(double* input, int length){
    input[(length/2-1)*length+(length/2-1)] = 1000;
    input[(length/2)*length+(length/2-1)]   = 1000;
    input[(length/2-1)*length+(length/2)]   = 1000;
    input[(length/2)*length+(length/2)]     = 1000;
}
// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */
    //Alloc Space in GPU
    int size = length * length * sizeof(double);
    int threadsPerBlockSide = 8;
    double blocksPerGridSide = ceil((length - 2) / threadsPerBlockSide);
    dim3 threadsPerBlock(threadsPerBlockSide, threadsPerBlockSide);
    dim3 nbBlocks(blocksPerGridSide, blocksPerGridSide);
    double* d_input;
    hipMalloc(&d_input, size);
    double* d_output;
    hipMalloc(&d_output, size);
    hipEventRecord(cpy_H2D_start);
    /* Copying array from host to device goes here */
    //Copy the array to the GPU
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);
    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    //Copy array from host to device
    hipEventRecord(comp_start);
    /* GPU calculation goes here */
    double *temp;
    hipDeviceSynchronize();
    for(int n=0; n<(int) iterations; n++)
    {
        single_iteration<<<nbBlocks,threadsPerBlock>>>(d_input, d_output, length);
        init_center<<<1,1>>>(d_output, length);
        hipDeviceSynchronize();
        temp = d_input;
        d_input = d_output;
        d_output = temp;
    }
    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */
    //Copy result array in CPU
    hipMemcpy(output, d_input, size, hipMemcpyDeviceToHost); // The result is stored at the end of each iteration in the input array
    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */
    hipFree(d_input);
    hipFree(d_output);

    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}